#include <iostream>
#include <hip/hip_runtime.h>

__globa__ void AddIntsCUDA(int *a, int *b)
{
    a[0] += b[0];
}

int main(int argc, char *argv[])
{
    int a = 5, b = 9;
    int *ptr_a, *ptr_b;

    hipMalloc(&ptr_a, sizeof(int));
    hipMalloc(&ptr_b, sizeof(int));

    hipMemcpy(ptr_a, &a, sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(ptr_b, &b, sizeof(int), hipMemcpyHostToDevice);

    AddIntsCUDA<<<1, 1>>>(ptr_a, ptr_b);

    hipMemcpy(&a, ptr_a, sizeof(int), hipMemcpyDeviceToHost);

    std::cout << "The answer is" << std::endl;

    hipFree(ptr_a);
    hipFree(ptr_b);

    return 0;
}